#include "hip/hip_runtime.h"
#include <tuple>
#include <limits>
#include "CheckError.cuh"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include "kernelBasic.hpp"

#define BLOCK_SIZE 32

const auto INF = std::numeric_limits<float>::infinity();

__global__ void computation(float* matrix, int k, int n){

  int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

  const unsigned int kj = k*n + j;
  const unsigned int ij = i*n + j;
  const unsigned int ik = i*n + k;

  if(i < n && j < n &&
          matrix[ik] != INF &&
          matrix[kj] != INF &&
          matrix[ik] + matrix[kj] < matrix[ij]) {
            matrix[ij] = matrix[ik] + matrix[kj];
      }
  }

namespace kernelBasic {

float kernelBasic(float* matrix, int n){

	float* matrix_device;

  hipEvent_t start, stop;

	hipError_t err = hipMalloc(&matrix_device, n * n * sizeof(float));

  if(hipSuccess != err){
    printf("error\n");
  }

	float msTime = 0;

	//copio i dati dall'host al deivce
	err = hipMemcpy(matrix_device, matrix, n * n * sizeof(float), hipMemcpyHostToDevice);

  if(hipSuccess != err){
    printf("error\n");
  }

  //definisco i blocchi
dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
dim3 num_blocks(ceil(n/((float)BLOCK_SIZE) + 1), ceil(n/((float)BLOCK_SIZE) + 1), 1);

hipEventCreate(&start);
hipEventCreate(&stop);

float millis =0;
hipEventRecord(start);
	//faccio eseguire i calcoli
	for(int i = 0; i < n; i++){
	   computation<<<num_blocks, block_size >>> (matrix_device, i, n);
  }

  hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);
	float timingCompute = millis/1000;

	std::cout << "TIMING COMPUTE: " << timingCompute <<  "s\n";


	hipMemcpy(matrix, matrix_device, n * n * sizeof(float), hipMemcpyDeviceToHost);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(matrix_device);

	return timingCompute;
	}
}
